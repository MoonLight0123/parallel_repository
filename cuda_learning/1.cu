#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */

  firstParallel<<<3,3>>>();
hipDeviceSynchronize();
  /*
   * Some code is needed below so that the CPU will wait
   * for the GPU kernels to complete before proceeding.
   */

}

#include <stdio.h>

__global__ void printSuccessForCorrectExecutionConfiguration()
{

  if(threadIdx.x == 1023 && blockIdx.x == 255)
  {
    printf("Success!\n");
  }
}

int main()
{
  /*
   * This is one possible execution context that will make
   * the kernel launch print its success message.
   */

  printSuccessForCorrectExecutionConfiguration<<<256, 1024>>>();

  /*
   * Don't forget kernel execution is asynchronous and you must
   * sync on its completion.
   */

  hipDeviceSynchronize();
}